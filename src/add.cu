#include "hip/hip_runtime.h"
#include <torch/script.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "operator.h"

// cuda vector add
__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

torch::Tensor AddCUDA(torch::Tensor a, torch::Tensor b, torch::Tensor c)
{
    int n = a.size(0);
    int block_size = 1024;
    int grid_size = (n + block_size - 1) / block_size;
    vector_add<<<grid_size, block_size>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), n);
    return c;
}